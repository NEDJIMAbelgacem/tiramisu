
#include <hip/hip_runtime.h>
#include <stdint.h>
static __global__ void _kernel_0(int32_t c1, double *buf_C_r)
{
	const int32_t __bx__ = (blockIdx.x + 0);
	const int32_t __tx__ = (threadIdx.x + 0);
	for (int32_t c7 = 0; (c7 <= 3); (c7 += 1))
	{
		for (int32_t c9 = 0; (c9 <= 45); (c9 += 1))
		{
			for (int32_t c11 = 0; (c11 <= 3); (c11 += 1))
			{
				for (int32_t c13 = 0; (c13 <= 45); (c13 += 1))
				{
					buf_C_r[(((((((0 + (c13 * 1)) + (c11 * (1 * 46))) + (c9 * ((1 * 46) * 4))) + (c7 * (((1 * 46) * 4) * 46))) + (__tx__ * ((((1 * 46) * 4) * 46) * 4))) + (__bx__ * (((((1 * 46) * 4) * 46) * 4) * 2))) + (c1 * ((((((1 * 46) * 4) * 46) * 4) * 2) * 64)))] = 0;
				};
			};
		};
	};
};
extern "C" int32_t _kernel_0_wrapper(int32_t c1, double *buf_C_r)
{
	{
		dim3 blocks((63 + 1), 1, 1);
		dim3 threads((1 + 1), 1, 1);
		_kernel_0<<<blocks, threads>>>(c1, buf_C_r);
	};
	return 0;
};
static __global__ void _kernel_1(int32_t c1, double *buf_C_i)
{
	const int32_t __bx__ = (blockIdx.x + 0);
	const int32_t __tx__ = (threadIdx.x + 0);
	for (int32_t c7 = 0; (c7 <= 3); (c7 += 1))
	{
		for (int32_t c9 = 0; (c9 <= 45); (c9 += 1))
		{
			for (int32_t c11 = 0; (c11 <= 3); (c11 += 1))
			{
				for (int32_t c13 = 0; (c13 <= 45); (c13 += 1))
				{
					buf_C_i[(((((((0 + (c13 * 1)) + (c11 * (1 * 46))) + (c9 * ((1 * 46) * 4))) + (c7 * (((1 * 46) * 4) * 46))) + (__tx__ * ((((1 * 46) * 4) * 46) * 4))) + (__bx__ * (((((1 * 46) * 4) * 46) * 4) * 2))) + (c1 * ((((((1 * 46) * 4) * 46) * 4) * 2) * 64)))] = 0;
				};
			};
		};
	};
};
extern "C" int32_t* _kernel_1_wrapper(int32_t c1, double *buf_C_i)
{
	{
		dim3 blocks((63 + 1), 1, 1);
		dim3 threads((1 + 1), 1, 1);
		_kernel_1<<<blocks, threads>>>(c1, buf_C_i);
	};
	return 0;
}